#include <stdio.h>
#include "util.h"

void check(hipError_t result, char const *const func, const char *const file,
           int const line) {
  if (result) {
    fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), hipGetErrorString(result), func);
    exit(EXIT_FAILURE);
  }
}

inline void __getLastCudaError(const char *errorMessage, const char *file,
                               const int line) {
  hipError_t err = hipGetLastError();

  if (hipSuccess != err) {
    fprintf(stderr,
            "%s(%i) : getLastCudaError() CUDA error :"
            " %s : (%d).\n",
            file, line, errorMessage, static_cast<int>(err));
    exit(EXIT_FAILURE);
  }
}

unsigned long int compute_interval_ms(struct timespec *start, struct timespec *end){
  return (end->tv_sec - start->tv_sec) * 1000 + (end->tv_nsec - start->tv_nsec) / 1000000;

}
