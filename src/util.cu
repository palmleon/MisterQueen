#include <stdio.h>
#include "util.h"

void check(hipError_t result, char const *const func, const char *const file,
           int const line) {
  if (result) {
    fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), hipGetErrorString(result), func);
    exit(EXIT_FAILURE);
  }
}

inline void __getLastCudaError(const char *errorMessage, const char *file,
                               const int line) {
  hipError_t err = hipGetLastError();

  if (hipSuccess != err) {
    fprintf(stderr,
            "%s(%i) : getLastCudaError() CUDA error :"
            " %s : (%d).\n",
            file, line, errorMessage, static_cast<int>(err));
    exit(EXIT_FAILURE);
  }
}
