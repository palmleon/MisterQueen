#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <sys/time.h>
#include "search.h"
#include "config.h"
#include "eval.h"
#include "gen.h"
#include "gpu.h"
#include "move.h"
#include "util.h"

#define LEN_POSITIONS 3

void sort_moves(Board *board, Move *moves, int count) {
    int best = -INF, index;
    for (int i = 0; i < count; i++) {
        Move *move = &(moves[i]);
        int score = score_move(board, move);
        if (score > best) {
            best = score;
            index = i;
        }
    }
    if (count >= 1) {
        Move tmp;
        tmp = moves[index];
        moves[index] = moves[0];
        moves[0] = tmp;
    }
}

int initial_sort_moves_rec(Board *board, int *positions, int len, int ply, int alpha, int beta) {
    int result;
    if (is_illegal(board)) {
        result = INF;
    }
    else if (len <= 0) {
        result = evaluate(board);
    }
    else {
        Undo undo;
        Move moves[MAX_MOVES];
        int count = gen_moves(board, moves);
        int *best_indexes = (int*) malloc(sizeof(int)*(len-1));
        int can_move = 0;
        for (int i = 0; i < count; i++) {
            Move *move = &moves[i];
            do_move(board, move, &undo);
            int score = -initial_sort_moves_rec(board, best_indexes, len - 1, ply + 1, -beta, -alpha);
            undo_move(board, move, &undo);
            if (score > -INF) {
                can_move = 1;
            }
            if (score >= beta) {
                return beta;
            }
            if (score > alpha) {
                alpha = score;
                positions[0] = i;
                for (int j = 1; j < len; j++) {
                    positions[j] = best_indexes[j-1];
                }
            }
        }
        result = alpha;
        if (!can_move) {
            if (is_check(board)) {
                result = -MATE + ply;
            } else {
                result = 0;
            }
        }
        free(best_indexes);
    }
    return result;
}

void initial_sort_moves(Board *board, Move *moves, int count, int *positions, int len) {
    Undo undo;
    int *best_indexes = (int*) malloc(sizeof(int)*(len-1));
    int best_score = -INF;
    for (int i = 0; i < count; i++) {
        Move *move = &(moves[i]);
        do_move(board, move, &undo);
        int score = -initial_sort_moves_rec(board, best_indexes, len-1, 1, -INF, +INF);
        undo_move(board, move, &undo);
        if (score > best_score) {
            best_score = score;
            positions[0] = i;
            for (int j = 1; j < len; j++) {
                positions[j] = best_indexes[j-1];
            }
        }
    }
    if (count >= 1){
        Move tmp;
        tmp = moves[positions[0]];
        moves[positions[0]] = moves[0];
        moves[0] = tmp;
    }
    free(best_indexes);
}

int alpha_beta_cpu(Board *board, int depth, int ply, int alpha, int beta, int *positions, int len_positions) {
    int result;
    if (is_illegal(board)) {
        result = INF;
    }
    else if (depth <= 0) {
        result = evaluate(board); 
    }
    else {
        Undo undo;
        Move moves[MAX_MOVES];
        int count = gen_moves(board, moves);
        int *scores = (int*) malloc (count * sizeof(int));
        int can_move = 0;
        if (ply < len_positions) {
            Move tmp;
            tmp = moves[positions[ply]];
            moves[positions[ply]] = moves[0];
            moves[0] = tmp;
        }
        else {
            sort_moves(board, moves, count);
        }
        if (count >= 1){
            do_move(board, &(moves[0]), &undo);
            int score = -alpha_beta_cpu(board, depth - 1, ply + 1, -beta, -alpha, positions, len_positions);
            undo_move(board, &(moves[0]), &undo);
            if (score > -INF) {
                can_move = 1;
            }
            if (score >= beta) {
                return beta;
            }
            if (score > alpha) {
                alpha = score;
            }
        }
        if (count > 1){ 
            Board *d_board;
            Move *d_moves;
            int *d_scores;
            checkCudaErrors(hipMalloc(&d_board, sizeof(Board)));
            checkCudaErrors(hipMalloc(&d_moves, count * sizeof(Move)));
            checkCudaErrors(hipMalloc(&d_scores, count * sizeof(int)));
            checkCudaErrors(hipMemcpy(d_board, board, sizeof(Board), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_moves, moves, count * sizeof(Move), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(d_scores, scores, count * sizeof(int), hipMemcpyHostToDevice));
            alpha_beta_gpu_kernel<<<count-1, dim3(1, THREADS_PER_NODE, 1), 64 * (sizeof(bb) + sizeof(int))>>>(d_board, depth - 1, -beta, -alpha, d_moves, d_scores); // first move already counted
            checkCudaErrors(hipMemcpy(scores, d_scores, count * sizeof(int), hipMemcpyDeviceToHost));
            hipFree(d_board); hipFree(d_moves); hipFree(d_scores);
            for (int i = 1; i < count; i++){
                if (scores[i] > -INF) {
                    can_move = 1;
                }
                if (scores[i] >= beta) {
                    return beta;
                }
                if (scores[i] > alpha) {
                    alpha = scores[i];
                }
            }
        }
        result = alpha;
        if (!can_move) {
            if (is_check(board)) {
                result = -MATE + ply;
            } else {
                result = 0;
            }
        }
    }
    return result;
}

int root_search(Board *board, int depth, int ply, int alpha, int beta, Move *result) {
    Undo undo;
    Move moves[MAX_MOVES];
    int positions[LEN_POSITIONS];
    int count = gen_moves(board, moves);
    int *scores = (int*) malloc (count * sizeof(int));
    initial_sort_moves(board, moves, count, positions, LEN_POSITIONS);
    for (int i = 0; i < count; i++){
        moves[i].already_executed = 0; //search has not begun yet!
    }
    Move *best = NULL;
    if (count >= 1){
        do_move(board, &(moves[0]), &undo);
        int score = -alpha_beta_cpu(board, depth - 1, ply + 1, -beta, -alpha, positions, LEN_POSITIONS);
        undo_move(board, &(moves[0]), &undo);
        if (score > alpha) {
            alpha = score;
            best = &(moves[0]);
        }
    }
    if (count > 1){ // da cambiare con la chiamata al kernel
        Board *d_board;
        Move *d_moves;
        int *d_scores;
        checkCudaErrors(hipMalloc(&d_board, sizeof(Board)));
        checkCudaErrors(hipMalloc(&d_moves, count * sizeof(Move)));
        checkCudaErrors(hipMalloc(&d_scores, count * sizeof(int)));
        checkCudaErrors(hipMemcpy(d_board, board, sizeof(Board), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_moves, moves, count * sizeof(Move), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_scores, scores, count * sizeof(int), hipMemcpyHostToDevice));
        alpha_beta_gpu_kernel<<<count-1, dim3(1, THREADS_PER_NODE, 1),  64 * (sizeof(bb)  + sizeof(int))>>>(d_board, depth - 1, -beta, -alpha, d_moves, d_scores);
        checkCudaErrors(hipMemcpy(scores, d_scores, count * sizeof(int), hipMemcpyDeviceToHost));
        hipFree(d_board); hipFree(d_moves); hipFree(d_scores);
        for (int i = 1; i < count; i++){
            if (scores[i] > alpha) {
                alpha = scores[i];
                best = &(moves[i]);
            }   
        }
    }
    if (best) {
        memcpy(result, best, sizeof(Move));
    }
    return alpha;
}

int do_search(Board *board, int uci, Move *move) {
    struct timespec start, end;
    int result = 1;
    int score = 0;
    const int depth = MAX_DEPTH;
    int lo = INF;
    int hi = INF;
    int alpha = score - lo;
    int beta = score + hi;
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    score = root_search(board, depth, 0, alpha, beta, move);
    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    if (uci) {
        char move_string[16];
        move_to_string(move, move_string);
        int millis = (end.tv_sec - start.tv_sec) * 1000 + (end.tv_nsec - start.tv_nsec) / 1000000;
        printf("Stats:\n| depth: %d\n| score: %d\n| time: %d ms\n",
               depth, score, millis);
        notate_move(board, move, move_string);
        printf("| best move: %s\n", move_string);
    }
    return result;

}