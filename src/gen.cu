#include "hip/hip_runtime.h"
#include <string.h>
#include <math.h>
#include "gen.h"


#define EMIT_MOVE(m, a, b) \
    (m)->src = (a); \
    (m)->dst = (b); \
    (m)->promotion = EMPTY; \
    (m)->already_executed = 0; \
    (m)++;

#define EMIT_PROMOTION(m, a, b, p) \
    (m)->src = (a); \
    (m)->dst = (b); \
    (m)->promotion = (p); \
    (m)->already_executed = 0; \
    (m)++;

#define EMIT_PROMOTIONS(m, a, b) \
    EMIT_PROMOTION(m, a, b, QUEEN) \
    EMIT_PROMOTION(m, a, b, ROOK) \
    EMIT_PROMOTION(m, a, b, BISHOP) \
    EMIT_PROMOTION(m, a, b, KNIGHT)

#define THREADS_PER_NODE 64

/* To verify if the current player is in check, 
   we generate all the opponent moves 
   and verify if any of them can directly attack the king
   color: player that may be in check
*/
__device__ __host__ int is_check(Board *board, char color){
    // for black, board->color >> 4 = 0x01
    // for white, board->color >> 4 = 0x00
    const int color_bit = (color ^ BLACK) >> 3;
    // coeff = -1 for white, +1 for black
    //const int coeff[2] = {-1, 1};
    const bb players_pieces[2] = {board->white, board->black}; // array defined to avoid an if-else
    const bb front_right_mask[2] = {0xfefefefefefefefeL, 0x7f7f7f7f7f7f7f7fL};
    const bb front_left_mask[2] = {0x7f7f7f7f7f7f7f7fL, 0xfefefefefefefefeL};
    const bb own_pieces = players_pieces[color_bit];
    const bb opponent_pieces = players_pieces[color_bit ^ 1];
    bb mask = ~own_pieces;
    const bb mask_pawn = opponent_pieces | board->ep;
    const bb mask_pawn_opp = own_pieces | board->ep;
    bb dsts = 0;

    for(int sq = 0; sq < 64; sq++){
        //char piece = board->squares[sq];
        char piece = board_get_piece(board, sq);
        if (COLOR(piece) >> 3 == color_bit){
            switch(PIECE(piece)){
                case PAWN: {
                    bb pawn_bb = BIT(sq);
                    bb a1 = pawn_bb & front_right_mask[color_bit];
                    bb a1_vec[2] = {a1 << 7, a1 >> 7};
                    a1 = a1_vec[color_bit] & mask_pawn;
                    bb a2 = pawn_bb & front_left_mask[color_bit];
                    bb a2_vec[2] = {a2 << 9, a2 >> 9};
                    a2 = a2_vec[color_bit] & mask_pawn;
                    dsts |= a1;
                    dsts |= a2;      
                }              
                    break;
                case KNIGHT:
                    #ifdef __CUDA_ARCH__
                    dsts |= d_BB_KNIGHT[sq] & mask;
                    #else
                    dsts |= BB_KNIGHT[sq] & mask;
                    #endif
                    break;
                case BISHOP:
                    dsts |= bb_bishop(sq, board->all) & mask;
                    break;
                case ROOK:
                    dsts |= bb_rook(sq, board->all) & mask;
                    break;
                case QUEEN:
                    dsts |= bb_queen(sq, board->all) & mask;
                    break;
                case KING:
                    #ifdef __CUDA_ARCH__
                    dsts |= d_BB_KING[sq] & mask;
                    #else
                    dsts |= BB_KING[sq] & mask;
                    #endif
                    break;
                default: // empty piece
                    break;
            }
        }
    }
    return (dsts & opponent_pieces & board->kings) != (long long) 0;
}

__device__ __host__ int is_illegal(Board *board){
    return is_check(board, board->color ^ BLACK);
}

/*
 * Move generation algorithm
 * To make it as parallelizable as possible, it has been conceived as an iteration
 * on all the squares of the board. If there is a piece on the board of the same color
 * as of the current player (i.e. who must move), then I generate all the possible moves
 * for that piece.
 * Since the Board representation is based on bitboards, it is possible to compute
 * all the possible moves using tables, built at initialization time.
 * Each of these tables is a bitmap, representing where it is possible to move
 * To better understand the tables, read the bb_init() function in bb.c
 */
__device__ __host__ int gen_moves(Board *board, Move *moves){
    Move *ptr = moves;
    // for black, board->color >> 4 = 0x01
    // for white, board->color >> 4 = 0x00
    //const int color_bit = board->color >> 4;
    //const int color_bit = board->color >> 3;
    const int color_bit = board->color / 8;
    // coeff = -1 for white, +1 for black
    //const int coeff[2] = {-1, 1};
    const bb players_pieces[2] = {board->white, board->black}; // array defined to avoid an if-else
    const bb promo[2] = {0xff00000000000000L, 0x00000000000000ffL}; // representation of the promotion rank
    const bb third_rank[2] = {0x0000000000ff0000L, 0x0000ff0000000000L}; // used for initial double move of pawn
    const bb front_right_mask[2] = {0xfefefefefefefefeL, 0x7f7f7f7f7f7f7f7fL};
    const bb front_left_mask[2] = {0x7f7f7f7f7f7f7f7fL, 0xfefefefefefefefeL};
    const bb own_pieces = players_pieces[color_bit];
    const bb opponent_pieces = players_pieces[color_bit ^ 1];
    bb mask = ~own_pieces;
    const bb mask_pawn = opponent_pieces | board->ep;
    const bb mask_pawn_opp = own_pieces | board->ep;
    const char castles[4] = {CASTLE_WHITE_KING, CASTLE_WHITE_QUEEN, CASTLE_BLACK_KING, CASTLE_BLACK_QUEEN};
    const bb castle_masks_1[4] = {0x0000000000000060L, 0x000000000000000eL, 0x6000000000000000L, 0x0e00000000000000L};
    const bb castle_masks_2[4] = {0x0000000000000070L, 0x000000000000001cL, 0x7000000000000000L, 0x1c00000000000000L};
    const int castle_king_pos_before[2] = {4, 60};
    const int castle_king_pos_after[4] = {6, 2, 62, 58};

    for(int sq = 0; sq < 64; sq++){
        //char piece = board->squares[sq];
        char piece = board_get_piece(board, sq);
        bb dsts = 0;
        // move a piece only if it is of the current moving player!
        if (COLOR(piece) == board->color){
            bb pawn_bb;
            switch(PIECE(piece)){
                case PAWN: {
                    pawn_bb = BIT(sq);
                    bb p1_vec[2] = {pawn_bb << 8, pawn_bb >> 8};
                    bb p1 = p1_vec[color_bit] & ~board->all;
                    bb p2 = p1 & third_rank[color_bit];
                    bb p2_vec[2] = {p2 << 8, p2 >> 8};
                    p2 = p2_vec[color_bit] & ~board->all;
                    bb a1 = pawn_bb & front_right_mask[color_bit];
                    bb a1_vec[2] = {a1 << 7, a1 >> 7};
                    a1 = a1_vec[color_bit] & mask_pawn;
                    bb a2 = pawn_bb & front_left_mask[color_bit];
                    bb a2_vec[2] = {a2 << 9, a2 >> 9};
                    a2 = a2_vec[color_bit] & mask_pawn;
                    dsts |= p1;
                    dsts |= p2;
                    dsts |= a1;
                    dsts |= a2;
                    }
                    break;
                case KNIGHT:
                    #ifdef __CUDA_ARCH__
                    dsts |= d_BB_KNIGHT[sq] & mask;
                    #else
                    dsts |= BB_KNIGHT[sq] & mask;
                    #endif
                    break;
                case BISHOP:
                    dsts = bb_bishop(sq, board->all) & mask;
                    break;
                case ROOK:
                    dsts = bb_rook(sq, board->all) & mask;
                    break;
                case QUEEN:
                    dsts = bb_queen(sq, board->all) & mask;
                    break;
                case KING:
                    #ifdef __CUDA_ARCH__
                    dsts |= d_BB_KING[sq] & mask;
                    #else
                    dsts |= BB_KING[sq] & mask;
                    #endif
                    break;
                default: // empty piece
                    break;
            }
            // Emit all the moves
            while (dsts) {
                int dst;
                POP_LSB(dst, dsts);
                if ((PIECE(piece) == PAWN) && (BIT(dst) & promo[color_bit])){
                    EMIT_PROMOTIONS(moves, sq, dst);
                }
                else {
                    EMIT_MOVE(moves, sq, dst);
                }
            }
        }
    }
    // GENERATE CASTLES
    mask = ~opponent_pieces;
    // look for opponent attacks in the squares where the king should move
    bb dsts = 0;
    if ((board->castle & castles[color_bit*2]) || board->castle & castles[color_bit*2+1]){
        for (int sq = 0; sq < 64; sq++){
            //char piece = board->squares[sq];
            char piece = board_get_piece(board, sq);
            if (COLOR(piece) != board->color){
                switch(PIECE(piece)){
                    case PAWN: {
                        bb pawn_bb = BIT(sq);
                        bb a1 = pawn_bb & front_right_mask[color_bit ^ 1];
                        bb a1_vec[2] = {a1 << 7, a1 >> 7};
                        a1 = a1_vec[color_bit^1] & mask_pawn_opp;
                        bb a2 = pawn_bb & front_left_mask[color_bit ^ 1];
                        bb a2_vec[2] = {a2 << 9, a2 >> 9};
                        a2 = a2_vec[color_bit^1] & mask_pawn_opp;
                        dsts |= a1 | a2;
                        }
                        break;
                    case KNIGHT:
                        #ifdef __CUDA_ARCH__
                        dsts |= d_BB_KNIGHT[sq] & mask;
                        #else
                        dsts |= BB_KNIGHT[sq] & mask;
                        #endif
                        break;
                    case BISHOP:
                        dsts |= bb_bishop(sq, board->all) & mask;
                        break;
                    case ROOK:
                        dsts |= bb_rook(sq, board->all) & mask;
                        break;
                    case QUEEN:
                        dsts |= bb_queen(sq, board->all) & mask;
                        break;
                    case KING:
                        #ifdef __CUDA_ARCH__
                        dsts |= d_BB_KING[sq] & mask;
                        #else
                        dsts |= BB_KING[sq] & mask;
                        #endif
                        break;
                    default: // empty piece
                        break;
                }
            }
        }
        for (int i = 0; i < 2; i++) {
            // check if the player can castle and, if that is the case,
            // where it can castle and whether there are pieces
            // between the king and the rook
            bb mask = castle_masks_2[color_bit*2+i];
            if ((board->castle & castles[color_bit*2+i])
                && (!(board->all & castle_masks_1[color_bit*2+i]))){
                    // if the opponent can only move to squares (dsts)
                    // which do not attack the king during castle (mask)
                    // emit the castle
                if (!(dsts & mask)) {
                    EMIT_MOVE(moves, castle_king_pos_before[color_bit], castle_king_pos_after[color_bit*2+i]);
                }
            }
        }
    }

    return moves - ptr; // incompatible with parallel code, for now it is just for refactoring
}


__global__ void gen_moves_gpu(Board *board_arr, Move *moves_arr, int *count_arr, int childIdx){
    
    const int idx = threadIdx.x;
    // declare different pointers to the same contiguous memory area: it is the only way to use the shared memory
    __shared__ bb dsts_array[64];
    __shared__ char pieces[64];
    //int *pieces = (int*) ((char*) dsts_array + 64*sizeof(bb));
    Board board = board_arr[childIdx];
    Move *moves = moves_arr + MAX_MOVES * childIdx;
    Move *ptr = moves;
    // for black, board->color >> 4 = 0x01
    // for white, board->color >> 4 = 0x00
    //const int color_bit = board->color >> 4;
    //const int color_bit = board->color >> 3;
    const int color_bit = board.color / 8;
    // coeff = -1 for white, +1 for black
    //const int coeff[2] = {-1, 1};
    const bb players_pieces[2] = {board.white, board.black}; // array defined to avoid an if-else
    const bb promo[2] = {0xff00000000000000L, 0x00000000000000ffL}; // representation of the promotion rank
    const bb third_rank[2] = {0x0000000000ff0000L, 0x0000ff0000000000L}; // used for initial double move of pawn
    const bb front_right_mask[2] = {0xfefefefefefefefeL, 0x7f7f7f7f7f7f7f7fL};
    const bb front_left_mask[2] = {0x7f7f7f7f7f7f7f7fL, 0xfefefefefefefefeL};
    const bb own_pieces = players_pieces[color_bit];
    const bb opponent_pieces = players_pieces[color_bit ^ 1];
    bb mask = ~own_pieces;
    const bb mask_pawn = opponent_pieces | board.ep;
    const bb mask_pawn_opp = own_pieces | board.ep;
    const char castles[4] = {CASTLE_WHITE_KING, CASTLE_WHITE_QUEEN, CASTLE_BLACK_KING, CASTLE_BLACK_QUEEN};
    const bb castle_masks_1[4] = {0x0000000000000060L, 0x000000000000000eL, 0x6000000000000000L, 0x0e00000000000000L};
    const bb castle_masks_2[4] = {0x0000000000000070L, 0x000000000000001cL, 0x7000000000000000L, 0x1c00000000000000L};
    const int castle_king_pos_before[2] = {4, 60};
    const int castle_king_pos_after[4] = {6, 2, 62, 58};
    bb dsts; 
    for(int sq = idx * (64 / THREADS_PER_NODE); sq < (idx + 1) * (64 / THREADS_PER_NODE); sq++){
        //char piece = board->squares[sq];
        //char piece = board_get_piece_gpu(board, sq);
        pieces[sq] = board_get_piece(&board, sq);
        char piece = pieces[sq];
        dsts = 0;
        //dsts_array[sq] = 0;
        // move a piece only if it is of the current moving player!
        if (COLOR(piece) == board.color){
            bb pawn_bb;
            switch(PIECE(piece)){
                case PAWN: {
                    pawn_bb = BIT(sq);
                    bb p1_vec[2] = {pawn_bb << 8, pawn_bb >> 8};
                    bb p1 = p1_vec[color_bit] & ~board.all;
                    bb p2 = p1 & third_rank[color_bit];
                    bb p2_vec[2] = {p2 << 8, p2 >> 8};
                    p2 = p2_vec[color_bit] & ~board.all;
                    bb a1 = pawn_bb & front_right_mask[color_bit];
                    bb a1_vec[2] = {a1 << 7, a1 >> 7};
                    a1 = a1_vec[color_bit] & mask_pawn;
                    bb a2 = pawn_bb & front_left_mask[color_bit];
                    bb a2_vec[2] = {a2 << 9, a2 >> 9};
                    a2 = a2_vec[color_bit] & mask_pawn;
                    dsts |= p1;
                    dsts |= p2;
                    dsts |= a1;
                    dsts |= a2;
                    //dsts_array[sq] |= p1;
                    //dsts_array[sq] |= p2;
                    //dsts_array[sq] |= a1;
                    //dsts_array[sq] |= a2;
                    }
                    break;
                case KNIGHT:
                    #ifdef __CUDA_ARCH__
                    //dsts_array[sq] |= d_BB_KNIGHT[sq] & mask;
                    dsts |= d_BB_KNIGHT[sq] & mask;
                    #else
                    //dsts |= BB_KNIGHT[sq] & mask;
                    #endif
                    break;
                case BISHOP:
                    //dsts_array[sq] = bb_bishop(sq, board->all) & mask;
                    dsts = bb_bishop(sq, board.all) & mask;
                    break;
                case ROOK:
                    //dsts_array[sq] = bb_rook(sq, board->all) & mask;
                    dsts = bb_rook(sq, board.all) & mask;
                    break;
                case QUEEN:
                    //dsts_array[sq] = bb_queen(sq, board->all) & mask;
                    dsts = bb_queen(sq, board.all) & mask;
                    break;
                case KING:
                    #ifdef __CUDA_ARCH__
                    //dsts_array[sq] |= d_BB_KING[sq] & mask;
                    dsts |= d_BB_KING[sq] & mask;                    
                    #else
                    //dsts |= BB_KING[sq] & mask;
                    #endif
                    break;
                default: // empty piece
                    break;
            }
        }
        // define the dsts for square sq in shared memory
        dsts_array[sq] = dsts;
    }
    __syncthreads();
    if (idx == 0) {
        for (int sq = 0; sq < 64; sq++) {
            // Emit all the moves
            dsts = dsts_array[sq];
            char piece = pieces[sq];
            //char piece = board_get_piece_gpu(board, sq);
            while (dsts) {
                int dst;
                POP_LSB(dst, dsts);
                if ((PIECE(piece) == PAWN) && (BIT(dst) & promo[color_bit])){
                    EMIT_PROMOTIONS(moves, sq, dst);
                }
                else {
                    EMIT_MOVE(moves, sq, dst);
                }
            }
        }
    }
    __syncthreads();

    // GENERATE CASTLES
    mask = ~opponent_pieces;
    // look for opponent attacks in the squares where the king should move
    //dsts = 0;
    if ((board.castle & castles[color_bit*2]) || board.castle & castles[color_bit*2+1]){
        //for (int sq = 0; sq < 64; sq++){
        for(int sq = idx * (64 / THREADS_PER_NODE); sq < (idx + 1) * (64 / THREADS_PER_NODE); sq++){
            //char piece = board_get_piece_gpu(board, sq);
            //dsts = 0;
            char piece = pieces[sq];
            if (COLOR(piece) != board.color){
                switch(PIECE(piece)){
                    case PAWN: {
                        bb pawn_bb = BIT(sq);
                        bb a1 = pawn_bb & front_right_mask[color_bit ^ 1];
                        bb a1_vec[2] = {a1 << 7, a1 >> 7};
                        a1 = a1_vec[color_bit^1] & mask_pawn_opp;
                        bb a2 = pawn_bb & front_left_mask[color_bit ^ 1];
                        bb a2_vec[2] = {a2 << 9, a2 >> 9};
                        a2 = a2_vec[color_bit^1] & mask_pawn_opp;
                        //dsts |= a1 | a2;
                        dsts = a1 | a2;
                        //dsts_array[sq] |= a1 | a2;
                        }
                        break;
                    case KNIGHT:
                        #ifdef __CUDA_ARCH__
                        //dsts |= d_BB_KNIGHT[sq] & mask;
                        dsts = d_BB_KNIGHT[sq] & mask;
                        //dsts_array[sq] |= d_BB_KNIGHT[sq] & mask;
                        #else
                        dsts = BB_KNIGHT[sq] & mask;
                        #endif
                        break;
                    case BISHOP:
                        dsts /*|*/= bb_bishop(sq, board.all) & mask;
                        //dsts_array[sq] |= bb_bishop(sq, board->all) & mask;
                        break;
                    case ROOK:
                        dsts /*|*/= bb_rook(sq, board.all) & mask;
                        //dsts_array[sq] |= bb_rook(sq, board->all) & mask;
                        break;
                    case QUEEN:
                        dsts /*|*/= bb_queen(sq, board.all) & mask;
                        //dsts_array[sq] |= bb_queen(sq, board->all) & mask;
                        break;
                    case KING:
                        #ifdef __CUDA_ARCH__
                        dsts /*|*/= d_BB_KING[sq] & mask;
                        //dsts_array[sq] |= d_BB_KING[sq] & mask;
                        #else
                        dsts |= BB_KING[sq] & mask;
                        #endif
                        break;
                    default: // empty piece
                        break;
                }
            }
            dsts_array[sq] = dsts;
        }
        __syncthreads();
        if (idx == 0) {
            for (int sq = 1; sq < 64; sq++) {
                dsts_array[0] |= dsts_array[sq];
            }
            for (int i = 0; i < 2; i++) {
                // check if the player can castle and, if that is the case,
                // where it can castle and whether there are pieces
                // between the king and the rook
                bb mask = castle_masks_2[color_bit*2+i];
                if ((board.castle & castles[color_bit*2+i])
                    && (!(board.all & castle_masks_1[color_bit*2+i]))){
                        // if the opponent can only move to squares (dsts)
                        // which do not attack the king during castle (mask)
                        // emit the castle
                    if (!(dsts_array[0] & mask)) {
                        EMIT_MOVE(moves, castle_king_pos_before[color_bit], castle_king_pos_after[color_bit*2+i]);
                    }
                }
            }
        }
    }
    if (idx == 0)
        count_arr[childIdx] = moves - ptr;
    return; // incompatible with parallel code, for now it is just for refactoring
}

int gen_legal_moves(Board *board, Move *moves) {
    Move *ptr = moves;
    Undo undo;
    Move temp[MAX_MOVES];
    int count = gen_moves(board, temp);
    for (int i = 0; i < count; i++) {
        Move *move = &temp[i];
        do_move(board, move, &undo);
        if (!is_illegal(board)) {
            memcpy(moves++, move, sizeof(Move));
        }
        undo_move(board, move, &undo);
    }
    return moves - ptr;
}
